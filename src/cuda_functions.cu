
#include <hip/hip_runtime.h>
/*****************************************************************
 *                        PINOCCHI0  V4.0                        *
 *  (PINpointing Orbit-Crossing Collapsed HIerarchical Objects)  *
 *****************************************************************
 
 This code was written by
 Pierluigi Monaco
 Copyright (C) 2016
 
 web page: http://adlibitum.oats.inaf.it/monaco/pinocchio.html
 
 This program is free software; you can redistribute it and/or modify
 it under the terms of the GNU General Public License as published by
 the Free Software Foundation; either version 2 of the License, or
 (at your option) any later version.
 
 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.
 
 You should have received a copy of the GNU General Public License
 along with this program; if not, write to the Free Software
 Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA
*/

#if defined(CUFFTMP)

#include "pinocchio.h"
#include <cuda_runtime.h>
#include <complex.h>
#include <cuComplex.h>
#include <cufftMp.h>

// -------------------------
// defines and variables
// -------------------------

//#define DEBUG

cufftDoubleReal        *real_grid;
cufftDoubleComplex *complex_grid;
cudaStream_t stream{};
//cufftHandle forward_plan;
//cufftHandle reverse_plan;
cudaLibXtDesc *forward_desc;
cudaLibXtDesc *reverse_desc;
MPI_Comm comm = MPI_COMM_WORLD;

#ifdef DEBUG
  FILE *results;
  char filename[300];
#endif

#define GRID MyGrids[ThisGrid]
#define int64 long long int

struct Box3D {
    int64 lower[3];
    int64 upper[3];
    int64 strides[3];
};

// -------------------------
// functions' prototypes
// -------------------------


//double greens_function                  (double *, double, int, int);
//int    cubes_order                      (const void *, const void *);

// -------------------------
// code segment
// -------------------------

/*
int cubes_order(const void *A, const void *B)
{
  float a = *(starts[*(int*)A]);
  float b = *(starts[*(int*)B]);

  return (a - b);

  /*
  if( a > b)
    return 1;
  if(a < b)
    return -1;
  return 0;
 
}
*/

/*
GIOVANNI
Redefine these functions involving FFT calls in a .cu extension file in which we can use
the NVIDIA cufftMP library. The reason is that cufftMP needs some C++ abstract descriptors
which don't have any counterpart in standard C
*/

/*
Write a function to assign a device to each MPI task
This may become redundant since it will be called by each function,
but for these first steps it's necessary so we know which task 
is using which GPU. This step is planned to be skipped in the future
*/

/*C++ extensions*/
auto make_box = [](int64 lower[3], int64 upper[3], int64 strides[3]) {
		  Box3D box;
		  for(int i = 0; i < 3; i++) {
		    box.lower[i] = lower[i];
		    box.upper[i] = upper[i];
		    box.strides[i] = strides[i];
		  }
		  return box;
		};


void cuda_init()
{
  int ndevices;
  cudaGetDeviceCount(&ndevices);
  cudaSetDevice(ThisTask % ndevices);

  if (ThisTask == 0)
    if (ndevices == 0){
      printf("No accelerators found!");
      return;
    }
}

  
int set_one_grid(int ThisGrid)
{

  cuda_init();
  
  cufftResult_t status; //Check whether cuda functions are returning
  
  //ptrdiff_t    alloc_local;
  //unsigned int pfft_flags;

  //Create the FFT cuda stream 
  //cudaStreamCreate(&stream);
  
  //Create the plans
  //cufftHandle forward_plan;
  //cufftHandle reverse_plan;

  /*
  status = cufftCreate(&GRID.forward_plan);
  if (status != CUFFT_SUCCESS) {printf("!!! forward plan cufftCreate ERROR %d !!!\n", status);}
  
  status = cufftCreate(&GRID.reverse_plan);
  if (status != CUFFT_SUCCESS) {printf("!!! reverse plan cufftCreate ERROR %d !!!\n", status);}
  */
  
  GRID.norm = (double)1.0 /
    ((double)GRID.Ntotal);
  
  GRID.CellSize = (double)GRID.BoxSize / GRID.GSglobal[_x_];

  /*Initialization*/
  int64 nx               = (int64)GRID.GSglobal[_x_];
  int64 ny               = (int64)GRID.GSglobal[_y_];
  int64 nz               = (int64)GRID.GSglobal[_z_];
  int64 nz_real          = nz;
  int64 nz_complex       = (((int64)(GRID.GSglobal[_z_])/2)+1);
  int64 nz_real_padded   = 2*nz_complex;

  Box3D box_real, box_complex;
  
  {
    // Input data are X-slabs. 
    // Strides are packed and in-place (i.e., real is padded)
    int64 lower[3]   = {nx / NTasks * (ThisTask),   0,  0};
    int64 upper[3]   = {nx / NTasks * (ThisTask+1), ny, nz_real};
    int64 strides[3] = {(upper[1]-lower[1])*nz_real_padded, nz_real_padded, 1};
    box_real = make_box(lower, upper, strides);
  }
  
  {
    // Output data are Y-slabs.
    // Strides are packed
    int64 lower[3]   = {0,  ny / NTasks * (ThisTask),   0};
    int64 upper[3]   = {nx, ny / NTasks * (ThisTask+1), nz_complex};
    int64 strides[3] = {(upper[1]-lower[1])*(upper[2]-lower[2]), (upper[2]-lower[2]), 1};
    box_complex = make_box(lower, upper, strides);
  }
  /*End of initialization*/
  
  //MPI_Comm comm = MPI_COMM_WORLD;
  
  //Create the FFT cuda stream and 
  //attach the MPI default communicator to the distributed FFT
  /*
  status = cufftMpAttachComm(GRID.forward_plan, CUFFT_COMM_MPI, &comm);
  if (status != CUFFT_SUCCESS) {printf("!!! cufftMpAttachComm ERROR %d !!!\n", status);}

  status = cufftSetStream(GRID.forward_plan, stream);
  if (status != CUFFT_SUCCESS) {printf("!!! cufftSetStream ERROR %d !!!\n", status);}

  status = cufftMpAttachComm(GRID.reverse_plan, CUFFT_COMM_MPI, &comm);
  if (status != CUFFT_SUCCESS) {printf("!!! cufftMpAttachComm ERROR %d !!!\n", status);}

  status = cufftSetStream(GRID.reverse_plan, stream);
  if (status != CUFFT_SUCCESS) {printf("!!! cufftSetStream ERROR %d !!!\n", status);}
    
  
  printf("[DBG CHECK START] %ld, %ld, %ld, %lld, %lld, %lld\n", GRID.GSstart[_x_], GRID.GSstart[_y_], GRID.GSstart[_z_],
	 real_lower[0], real_lower[1], real_lower[2]);
  printf("[DBG CHECK START_K] %ld, %ld, %ld, %lld, %lld, %lld\n", GRID.GSstart_k[_x_], GRID.GSstart_k[_y_], GRID.GSstart_k[_z_],
	 c_lower[0], c_lower[1], c_lower[2]);
  printf("[DBG CHECK LOCAL] %ld, %ld, %ld, %lld, %lld, %lld\n", GRID.GSlocal[_x_], GRID.GSlocal[_y_], GRID.GSlocal[_z_],
	 real_upper[0], real_upper[1], real_upper[2]);
  printf("[DBG CHECK LOCAL_K] %ld, %ld, %ld, %lld, %lld, %lld\n", GRID.GSlocal_k[_x_], GRID.GSlocal_k[_y_], GRID.GSlocal_k[_z_],
	 c_upper[0], c_upper[1], c_upper[2]);
  
  printf("Numbers: %lld, %lld\n", nz_complex, nz_real_padded);
  */

  /*
  //Distribute r2c plan
  status = cufftXtSetDistribution(GRID.forward_plan, 3, box_real.lower, box_real.upper, box_complex.lower,
				  box_complex.upper, box_real.strides, box_complex.strides);
  if (status != CUFFT_SUCCESS) {printf("!!! cufftXtSetDistribution r2c ERROR %d !!!\n", status);}

  //Distribute c2r plan
  status = cufftXtSetDistribution(GRID.reverse_plan, 3, box_real.lower, box_real.upper, box_complex.lower,
				  box_complex.upper, box_real.strides, box_complex.strides);
  if (status != CUFFT_SUCCESS) {printf("!!! cufftXtSetDistribution c2r ERROR %d !!!\n", status);}
  */
  //pfft_flags  = PFFT_MEASURE;
  //pfft_flags  = 0;
  //if(params.use_transposed_fft)
  // pfft_flags  |= PFFT_TRANSPOSED_OUT;

  /*
  alloc_local = pfft_local_size_dft_r2c_3d(GRID.GSglobal, FFT_Comm, pfft_flags,
					   GRID.GSlocal, GRID.GSstart,
					   GRID.GSlocal_k, GRID.GSstart_k);
  */
  
  dprintf(VDBG, ThisTask, "[set grid %02d] task %d %lld "
	  "i: %lld %lld %lld - i start: %lld %lld %lld - "
	  "o: %lld %lld %lld - o start: %lld %lld %lld\n",
	  ThisGrid, ThisTask, nz_complex,
	  box_real.upper[0], box_real.upper[1], box_real.upper[2],
	  box_real.lower[0], box_real.lower[1], box_real.lower[2],
	  box_complex.upper[0], box_complex.upper[1], box_complex.upper[2],
	  box_complex.lower[0], box_complex.lower[1], box_complex.lower[2]);
  
  
  GRID.total_local_size_fft = 2 * nz_complex;
  GRID.total_local_size     = nx * ny * nz_complex;
  
  MyGrids[0].off = 0;
  // order the sub-blocks by row-major order (i, j, k), k first, then j, then i

  /* int           i; */
  /* intint index; */

  /* starts         = (point*)malloc(sizeof(point) * NTasks); */
  
  /* MPI_Allgather(GRID.GSlocal, sizeof(point), MPI_BYTE, starts, sizeof(point), MPI_BYTE, MPI_COMM_WORLD); */
  /* for(i = 0; i < NTasks; i++) */
  /*   { */
  /*     index            = (starts[i][_x_]*GRID.GSglobal[_y_] + starts[i][_y_])*GRID.GSglobal[_z_] + starts[i][_z_]; */
  /*     starts[i][_x_]   = index; */
  /*     cubes_ordering[i] = i; */
  /*   } */

  /* qsort(cubes_ordering, NTasks, sizeof(int), cubes_order); */

  /* free(starts); */

  return 0;
}




int compute_fft_plans()
{
  cuda_init();
  //ptrdiff_t DIM[3];
  long long int DIM[3];
  //int       pfft_flags;
  int       ThisGrid;

  cufftResult_t status; //Check whether cuda functions are returning
    
  dprintf(VMSG, 0, "[%s] Computing fft plans\n",fdate());

  for (ThisGrid = 0; ThisGrid < Ngrids; ThisGrid++)
    {

      //cufftHandle GRID.forward_plan;
      //cufftHandle GRID.reverse_plan;
  
      cudaStreamCreate(&stream);
  
      status = cufftCreate(&GRID.forward_plan);
      if (status != CUFFT_SUCCESS) {printf("!!! forward plan cufftCreate ERROR %d !!!\n", status);}
  
      status = cufftCreate(&GRID.reverse_plan);
      if (status != CUFFT_SUCCESS) {printf("!!! reverse plan cufftCreate ERROR %d !!!\n", status);}
  
      status = cufftMpAttachComm(GRID.forward_plan, CUFFT_COMM_MPI, &comm);
      if (status != CUFFT_SUCCESS) {printf("!!! cufftMpAttachComm ERROR %d !!!\n", status);}

      status = cufftSetStream(GRID.forward_plan, stream);
      if (status != CUFFT_SUCCESS) {printf("!!! cufftSetStream ERROR %d !!!\n", status);}

      status = cufftMpAttachComm(GRID.reverse_plan, CUFFT_COMM_MPI, &comm);
      if (status != CUFFT_SUCCESS) {printf("!!! cufftMpAttachComm ERROR %d !!!\n", status);}

      status = cufftSetStream(GRID.reverse_plan, stream);
      if (status != CUFFT_SUCCESS) {printf("!!! cufftSetStream ERROR %d !!!\n", status);}

      /*Initialization*/
      int64 nx               = (int64)GRID.GSglobal[_x_];
      int64 ny               = (int64)GRID.GSglobal[_y_];
      int64 nz               = (int64)GRID.GSglobal[_z_];
      int64 nz_real          = nz;
      int64 nz_complex       = (((int64)(GRID.GSglobal[_z_])/2)+1);
      int64 nz_real_padded   = 2*nz_complex;

      Box3D box_real, box_complex;
  
      {
	// Input data are X-slabs. 
	// Strides are packed and in-place (i.e., real is padded)
	int64 lower[3]   = {nx / NTasks * (ThisTask),   0,  0};
	int64 upper[3]   = {nx / NTasks * (ThisTask+1), ny, nz_real};
	int64 strides[3] = {(upper[1]-lower[1])*nz_real_padded, nz_real_padded, 1};
	box_real = make_box(lower, upper, strides);
      }
  
      {
	// Output data are Y-slabs.
	// Strides are packed
	int64 lower[3]   = {0,  ny / NTasks * (ThisTask),   0};
	int64 upper[3]   = {nx, ny / NTasks * (ThisTask+1), nz_complex};
	int64 strides[3] = {(upper[1]-lower[1])*(upper[2]-lower[2]), (upper[2]-lower[2]), 1};
	box_complex = make_box(lower, upper, strides);
      }
      /*End of initialization*/



      
      //Distribute r2c plan
      status = cufftXtSetDistribution(GRID.forward_plan, 3, box_real.lower, box_real.upper, box_complex.lower,
				      box_complex.upper, box_real.strides, box_complex.strides);
      if (status != CUFFT_SUCCESS) {printf("!!! cufftXtSetDistribution r2c ERROR %d !!!\n", status);}

      //Distribute c2r plan
      status = cufftXtSetDistribution(GRID.reverse_plan, 3, box_real.lower, box_real.upper, box_complex.lower,
				      box_complex.upper, box_real.strides, box_complex.strides);
      if (status != CUFFT_SUCCESS) {printf("!!! cufftXtSetDistribution c2r ERROR %d !!!\n", status);}

      
      //Allocate the descriptors
      /*
      status = cufftXtMalloc(GRID.forward_plan, &forward_desc, CUFFT_XT_FORMAT_DISTRIBUTED_INPUT);
      if (status != CUFFT_SUCCESS) {printf("!!! cufftXtMalloc forward ERROR %d !!!\n", status);}

      status = cufftXtMalloc(GRID.reverse_plan, &reverse_desc, CUFFT_XT_FORMAT_DISTRIBUTED_OUTPUT);
      if (status != CUFFT_SUCCESS) {printf("!!! cufftXtMalloc reverse ERROR %d !!!\n", status);}
      */
      DIM[_x_] = GRID.GSglobal[_x_];
      DIM[_y_] = GRID.GSglobal[_y_];
      DIM[_z_] = GRID.GSglobal[_z_];

      // create plan for out-of-place DFT 

      //pfft_flags = PFFT_MEASURE | PFFT_TUNE;
      //pfft_flags = PFFT_MEASURE ;
      /*
      pfft_flags = 0;
      if(params.use_transposed_fft)
	pfft_flags |= PFFT_TRANSPOSED_OUT;
#ifdef USE_FFT_THREADS
      fftw_plan_with_nthreads(internal.nthreads_fft);
      pfft_plan_with_nthreads(internal.nthreads_fft);
#endif
      //GRID.forward_plan = pfft_plan_dft_r2c_3d(DIM, rvector_fft[ThisGrid], cvector_fft[ThisGrid],
      //				       FFT_Comm, PFFT_FORWARD, pfft_flags);
      */
      
      //Make the forward FFT plan
      
      size_t workspace_f;
      status = cufftMakePlan3d(GRID.forward_plan, DIM[_x_], DIM[_y_], DIM[_z_], CUFFT_D2Z, &workspace_f);
      if (status != CUFFT_SUCCESS) {printf("!!! cufftMakePlan3d ERROR %d !!!\n", status);}

      
      DIM[_x_] = GRID.GSglobal[_x_];
      DIM[_y_] = GRID.GSglobal[_y_];
      DIM[_z_] = GRID.GSglobal[_z_];
      
      //pfft_flags = PFFT_MEASURE | PFFT_TUNE;
      //pfft_flags = PFFT_MEASURE;
      /*
      pfft_flags = 0;
      if(params.use_transposed_fft)
	pfft_flags |= PFFT_TRANSPOSED_IN;
#ifdef USE_FFT_THREADS
      fftw_plan_with_nthreads(internal.nthreads_fft);
      pfft_plan_with_nthreads(internal.nthreads_fft);
#endif
      GRID.reverse_plan = pfft_plan_dft_c2r_3d(DIM, cvector_fft[ThisGrid], rvector_fft[ThisGrid], 
					       FFT_Comm, PFFT_BACKWARD, pfft_flags);
      */

      //Make the reverse FFT plan
      
      size_t workspace_r;
      status = cufftMakePlan3d(GRID.reverse_plan, DIM[_x_], DIM[_y_], DIM[_z_], CUFFT_Z2D, &workspace_r);
      if (status != CUFFT_SUCCESS) {printf("!!! cufftMakePlan3d ERROR %d !!!\n", status);}
      
    }
      
  dprintf(VMSG, 0, "[%s] fft plans done\n",fdate());

  return 0;
}


double forward_transform(int ThisGrid)
{
  cuda_init();
  double time;
  cufftDoubleComplex *complex_grid;
  cufftDoubleReal *real_grid;
  cudaError_t mmm;
  cufftResult_t status;


  //Define the descriptors
  cudaLibXtDesc *forward_desc;
  cudaLibXtDesc *reverse_desc;
  
  // Alloco fftwgrid su GPU utilizzando cudaMalloc
  /*
  long long int DIM[3];
  DIM[_x_] = GRID.GSglobal[_x_];
  DIM[_y_] = GRID.GSglobal[_y_];
  DIM[_z_] = GRID.GSglobal[_z_];

  size_t workspace_f;
  status = cufftMakePlan3d(GRID.forward_plan, DIM[_x_], DIM[_y_], DIM[_z_], CUFFT_D2Z, &workspace_f);
  if (status != CUFFT_SUCCESS) {printf("!!! cufftMakePlan3d ERROR %d !!!\n", status);}
  */
  long long unsigned size_finta_fft = (long long unsigned)MyGrids[0].total_local_size;

  mmm=cudaMalloc(&real_grid, (size_t)(size_finta_fft*sizeof(cufftDoubleReal)));
  if (mmm != cudaSuccess) {printf("!!! cudaMalloc real ERROR %d !!!\n", mmm);}
  
  mmm=cudaMalloc(&complex_grid, (size_t)(size_finta_fft*sizeof(cufftDoubleComplex)));
  if (mmm != cudaSuccess) {printf("!!! cudaMalloc complex ERROR %d !!!\n", mmm);}

  //Allocate the descriptors
     
  status = cufftXtMalloc(GRID.forward_plan, &forward_desc, CUFFT_XT_FORMAT_DISTRIBUTED_INPUT);
  if (status != CUFFT_SUCCESS) {printf("!!! cufftXtMalloc forward ERROR %d !!!\n", status);}

  status = cufftXtMalloc(GRID.reverse_plan, &reverse_desc, CUFFT_XT_FORMAT_DISTRIBUTED_OUTPUT);
  if (status != CUFFT_SUCCESS) {printf("!!! cufftXtMalloc reverse ERROR %d !!!\n", status);}
     
       
  time=MPI_Wtime();

  // Copy data from the CPU to the GPU.
  // The CPU data is distributed according to CUFFT_XT_FORMAT_DISTRIBUTED_INPUT
  status = cufftXtMemcpy(GRID.forward_plan, forward_desc, real_grid, CUFFT_COPY_HOST_TO_DEVICE);
  if (status != CUFFT_SUCCESS) {printf("!!! cufftXtMemcpyH2D ERROR %d !!!\n", status);}
  
  status = cufftXtExecDescriptor(GRID.forward_plan, forward_desc, forward_desc, CUFFT_FORWARD);
  if (status != CUFFT_SUCCESS) {printf("!!! cufftExecDescriptor ERROR %d !!!\n", status);}
  
  // Copy memory back to the CPU. Data is now distributed according to CUFFT_XT_FORMAT_DISTRIBUTED_OUTPUT
  status = cufftXtMemcpy(GRID.forward_plan, complex_grid, forward_desc, CUFFT_COPY_DEVICE_TO_HOST);
  if (status != CUFFT_SUCCESS) {printf("!!! cufftXtMemcpyD2H ERROR %d !!!\n", status);}
  
  return MPI_Wtime()-time;
}


double reverse_transform(int ThisGrid)
{
  cuda_init();
  int i;
  double time;
  cufftResult_t status;
  /*
  long long int DIM[3];
  DIM[_x_] = GRID.GSglobal[_x_];
  DIM[_y_] = GRID.GSglobal[_y_];
  DIM[_z_] = GRID.GSglobal[_z_];

  size_t workspace_f;
  status = cufftMakePlan3d(GRID.reverse_plan, DIM[_x_], DIM[_y_], DIM[_z_], CUFFT_D2Z, &workspace_f);
  if (status != CUFFT_SUCCESS) {printf("!!! cufftMakePlan3d ERROR %d !!!\n", status);}
  */
  
  time=MPI_Wtime();

  // Copy data from the CPU to the GPU.
  // The CPU data is distributed according to CUFFT_XT_FORMAT_DISTRIBUTED_INPUT
  status = cufftXtMemcpy(GRID.reverse_plan, reverse_desc, complex_grid, CUFFT_COPY_HOST_TO_DEVICE);
  if (status != CUFFT_SUCCESS) {printf("!!! cufftXtMemcpyH2D ERROR %d !!!\n", status);}
  
  status = cufftXtExecDescriptor(GRID.reverse_plan, reverse_desc, reverse_desc, CUFFT_INVERSE);
  if (status != CUFFT_SUCCESS) {printf("!!! cufftExecDescriptor ERROR %d !!!\n", status);}
  
  // Copy memory back to the CPU. Data is now distributed according to CUFFT_XT_FORMAT_DISTRIBUTED_OUTPUT
  status = cufftXtMemcpy(GRID.reverse_plan, real_grid, reverse_desc, CUFFT_COPY_DEVICE_TO_HOST);
  if (status != CUFFT_SUCCESS) {printf("!!! cufftXtMemcpyD2H ERROR %d !!!\n", status);}
  
  dvec         NORM   = {GRID.norm, GRID.norm, GRID.norm, GRID.norm};
  unsigned int mysize = GRID.total_local_size_fft / 4;

  rvector_fft = (double**)real_grid;
     
#pragma GCC ivdep  
  for (i = 0; i < mysize; i++)
    *((dvec*)rvector_fft[ThisGrid] + i) *= NORM;
    
  for (i = GRID.total_local_size_fft - GRID.total_local_size_fft%4 ; i < GRID.total_local_size_fft; i++)
    rvector_fft[ThisGrid][i] *= GRID.norm;

  // non-vector code 
  /* for (i = 0 ; i < GRID.total_local_size_fft; i++) */
  /*   rvector_fft[ThisGrid][i] *= GRID.norm; */

  return MPI_Wtime() - time;
}


int finalize_fft()
{
  cuda_init();
#ifndef RECOMPUTE_DISPLACEMENTS
  int ThisGrid;

  cufftResult_t status; //Check whether cuda functions are returning
  
  for (ThisGrid = Ngrids-1; ThisGrid >= 0; ThisGrid--)
    {
      cufftXtFree(forward_desc);
      cufftXtFree(reverse_desc);
      status = cufftDestroy(GRID.forward_plan);
      status = cufftDestroy(GRID.reverse_plan);
      if (status != CUFFT_SUCCESS) {printf("!!! cufftDestroy fftwgrid ERROR %d !!!\n", status);}
    }

  /* for (ThisGrid = Ngrids-1; ThisGrid >= 0; ThisGrid--) */
  /*   { */
  /*     pfft_free(GRID.forward_plan); */
  /*     pfft_free(GRID.reverse_plan); */
  /*   } */

  cudaStreamDestroy(stream);
  //pfft_cleanup();
  //MPI_Comm_free(&FFT_Comm);
#endif

  return 0;
}
#endif //close initial if defined(CUFFTMP)
